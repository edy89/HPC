#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

void initialData(int *ip, int size)
{
    srand (time(NULL));

    for (int i = 0; i < size; i++)
    {
        ip[i] = rand() % 2;
    }

    return;
}

__global__ void movimientoAutos(int *d_A, int *d_B, const int N)
{
    int a=0;
    int b=0;
    int x1=0;
    int y1=0;
    int u=0;
    int entrada;
    int entrada2;
    int i = threadIdx.x;

    if (i < N)
    {

        if(i==0){
                entrada  = 1;
                entrada2 = 1;
            }   
            else{
                entrada  = d_A[i-1];
                entrada2 = d_A[i];


            }

            if ( entrada==1 && entrada2==0)
             {
               d_B[i-1] = 0;
               d_B[i] = 1;
             }
            else{

                a = d_A[i];
                b = d_A[i+1];
                if     (a==0 && b==0){x1=0;y1=0;u=0;}
                else if(a==0 && b==1){x1=0;y1=1;u=0;}
                else if(a==1 && b==0){x1=0;y1=1;u=1;}
                else if(a==1 && b==1){x1=1;y1=1;u=0;}

                if (u==1)
                {
                    d_B[i]   = x1;
                    d_B[i+1] = y1;
                    
                }
                else{
                    d_B[i]   = x1;
                    d_B[i+1] = y1;
                        
             
                }


            }
    }            
        
}


int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);
    clock_t start_t, end_t;
    double duration;

    int parametro,x;
    parametro = atoi(argv[1]);
    x = parametro;


    int p,cantidadinter;
    p = atoi(argv[2]);
    cantidadinter = p;

    int dev = 0;
    hipSetDevice(dev);

    int nElem = x;
    printf("Vector size %d\n", nElem);

    size_t nBytes = nElem * sizeof(int);

    int *h_A, *h_B, *gpuRef;
    h_A     = (int *)malloc(nBytes);
    h_B     = (int *)malloc(nBytes);
    gpuRef  = (int *)malloc(nBytes);

    initialData(h_A, nElem);
/*    for (int i = 0; i < nElem; ++i)
    {
        printf("[%d]", h_A[i]);
    }*/


    memset(gpuRef,  0, nBytes);
    memset(h_B,  0, nBytes);


    int *d_A, *d_B; //*d_C;
    hipMalloc((int**)&d_A, nBytes);
    hipMalloc((int**)&d_B, nBytes);



    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);


    dim3 block (nElem);
    dim3 grid  (1);

    start_t = clock();
    for (int i = 0; i < cantidadinter; ++i)
    {
        movimientoAutos<<<grid, block>>>(d_A, d_B, nElem);       
        //printf("\nExecution configure <<<%d, %d>>>\n", grid.x, block.x);

        hipMemcpy(h_B, d_B, nBytes, hipMemcpyDeviceToHost);
        hipMemcpy(h_A, d_A, nBytes, hipMemcpyDeviceToHost);
        int a=0;
        int b=0;
        int u=0;
        int a1,b1;
        
        a=h_A[x-1];
        b=h_A[0];

        if     (a==0 && b==0){a1=0;b1=0;u=0;}
        else if(a==0 && b==1){a1=0;b1=1;u=0;}
        else if(a==1 && b==0){a1=0;b1=1;u=1;}
        else if(a==1 && b==1){a1=1;b1=1;u=0;}

        if(u==1)
        {
            h_B[x-1] = a1;
            h_B[0]   = b1;                  //guardamos los resultados en el result
        }

/*        for (int i = 0; i < nElem; ++i)
        {
           h_A[i] = h_B[i];
           printf("[%d]", h_B[i]);
        }

        printf("\n");*/

        hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    }
    end_t = clock();
    duration = (double)(end_t - start_t) / CLOCKS_PER_SEC;
    printf("\nEL tiempo fue: %f\n",duration);

/*    printf("----------------------\n");

    for (int i = 0; i < nElem; ++i)
    {
        printf("[%d]",h_B[i]);
    }

    printf("\n");*/

    hipFree(d_A);
    hipFree(d_B);


    free(h_A);
    free(h_B);
    free(gpuRef);

    hipDeviceReset();
    return(0);
}
