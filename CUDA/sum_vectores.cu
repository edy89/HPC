#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <chrono>
 
typedef std::chrono::high_resolution_clock Clock;
 
#define ITER 65535
 
// Version CPU de la función suma de vectores
void vector_add_cpu(int *a, int *b, int *c, int n) {
    int i;
 
    // Add the vector elements a and b to the vector c
    for (i = 0; i < n; ++i) {
    c[i] = a[i] + b[i];
    }
}
 
// Versión GPU de la función suma de vectores
__global__ void vector_add_gpu(int *gpu_a, int *gpu_b, int *gpu_c, int n) {
    int i = threadIdx.x;
    // No es necesario el loop for por que el runtime de CUDA
    // maneja estos hilos ITER veces
    gpu_c[i] = gpu_a[i] + gpu_b[i];
}
 
int main() {
 
    int *a, *b, *c;
    int *gpu_a, *gpu_b, *gpu_c;
 
    a = (int *)malloc(ITER * sizeof(int));
    b = (int *)malloc(ITER * sizeof(int));
    c = (int *)malloc(ITER * sizeof(int));
 
    // Necesitamos variables accesibles en CUDA,
    // para eso cudaMallocManaged nos las provee
    hipMallocManaged(&gpu_a, ITER * sizeof(int));
    hipMallocManaged(&gpu_b, ITER * sizeof(int));
    hipMallocManaged(&gpu_c, ITER * sizeof(int));
 
    for (int i = 0; i < ITER; ++i) {
        a[i] = i;
        b[i] = i;
        c[i] = i;
    }
 
    // Llama a la versión CPU y la temporiza
    auto cpu_start = Clock::now();
    vector_add_cpu(a, b, c, ITER);
    auto cpu_end = Clock::now();
    std::cout << "vector_add_cpu: "
    << std::chrono::duration_cast<std::chrono::nanoseconds>(cpu_end - cpu_start).count()
    << " nanoseconds.\n";
 
    // Llama a la versión GPU y la temporiza
    // Los triples <> es una extensión del runtime CUDA que permite
    // que los parametros de una llamada al kernel CUDA sean pasados
    // En este ejemplo estamos pasando un thread block con ITER threads
    auto gpu_start = Clock::now();
    vector_add_gpu <<<1, ITER>>> (gpu_a, gpu_b, gpu_c, ITER);
    hipDeviceSynchronize();
    auto gpu_end = Clock::now();
    std::cout << "vector_add_gpu: "
    << std::chrono::duration_cast<std::chrono::nanoseconds>(gpu_end - gpu_start).count()
    << " nanoseconds.\n";
 
    // Libere la memoria basada en la función GPU allocations
    hipFree(a);
    hipFree(b);
    hipFree(c);
 
    // Libere la memoria basada en la función CPU allocations
    free(a);
    free(b);
    free(c);
 
    return 0;
}