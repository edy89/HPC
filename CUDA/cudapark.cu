#include <stdio.h>
#include <stdlib.h>
#include <time.h>
# include <string.h>
#include <hiprand/hiprand.h>
#include <hip/hip_runtime.h>


__global__ void mover_autos(int *d_A, int *d_B, int x)
{
/*	int a=0;
	int b=0;
	int x1=0;
	int y1=0;
	int u=0;*/
	int i = threadIdx.x;


	if (i < x)
	{
		d_B[i] = 3;

/*		a=d_A[i];
		b=d_A[i+1];
		if 	   (a==0 && b==0){x1=0;y1=0;u=0;}
			else if(a==0 && b==1){x1=0;y1=1;u=0;}
			else if(a==1 && b==0){x1=0;y1=1;u=1;}
			else if(a==1 && b==1){x1=1;y1=1;u=0;}

	 		if (u==1)
	 		{
	 			d_B[i] = x1;
	 			d_B[i+1] = y1;	
	 			i++;
	 		}
	 		else{
	 			d_B[i]=x1;
	 			d_B[i+1]=y1;
	 		}*/
	}			 
		
/*	int a1,b1;
	a=d_A[x-1];
	b=d_A[0];
	if 	   (a==0 && b==0){a1=0;b1=0;u=0;}
	else if(a==0 && b==1){a1=0;b1=1;u=0;}
	else if(a==1 && b==0){a1=0;b1=1;u=1;}
	else if(a==1 && b==1){a1=1;b1=1;u=0;}

	if(u==1)
	{
		d_B[x-1]=a1;
		d_B[0]=b1;					//guardamos los resultados en el result
	}*/
}


int main(int argc, char *argv[])
{
	
    clock_t start_t, end_t;
    double duration;
    srand (time(NULL));

    
    int parametro,x;
    parametro = atoi(argv[1]);
    x = parametro;


	int p,cantidadinter;
    p = atoi(argv[2]);
    cantidadinter = p;
    //printf("%d\n", cantidadinter);

    size_t nBytes = x * sizeof(int);

	int* h_A = (int*)malloc(nBytes);
	int* h_B = (int*)malloc(nBytes);
	//int* h_C = (int*)malloc(nBytes);

	//int dev = 0;
    //cudaSetDevice(dev);

	// declare device vectors in the device (GPU) memory
	int *d_A,*d_B;//*d_C;

	for(int i = 0;i < x ; i++)
	 {
		h_A[i] = 1;
		//printf("[%d]", h_A[i]);
     } 

	printf("\n");

    hipMalloc((int **)&d_A, nBytes);
    hipMalloc((int **)&d_B, nBytes);

	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

	for (int i = 0; i < x; ++i)
	{
		printf("%d\n", h_A[i]);
		printf("%d\n", d_A[i]);
	}

	start_t = clock();

	for(int k=1;k<cantidadinter;k++)
	{
	  mover_autos <<<1, 4>>> (d_A, d_B, 4);
 	}

 	hipMemcpy(h_A, d_A, nBytes, hipMemcpyDeviceToHost);
	hipMemcpy(h_B, d_B, nBytes, hipMemcpyDeviceToHost);


    end_t = clock();
    duration = (double)(end_t - start_t) / CLOCKS_PER_SEC;
    printf("\nEL tiempo fue: %f",duration);
    hipFree(d_A);
  	hipFree(d_B);

	free (h_A);
	free(h_B);

	return hipDeviceReset();
	//return 0;
}